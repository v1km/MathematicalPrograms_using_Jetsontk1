//PROGRAM01
//THIS PROGRAM DEMONSTRATE SQUARING AN ARRAY USING A SIMPLE CUDA KERNEL
//WITH MEASURING CPU AND GPU TIME AND SPEED UP
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<time.h>

__global__ void SquareKernel(int *,int);
void SquareSerial(int*, int );

int main()
{
  int i; //loop variable
  int blockSize=128, blocks; //for cuda blocks
  hipError_t err;//for error checking in cuda API
  int size=200;
  /***********************************/
  float timespentCPU, timespentGPU;
  clock_t start1, stop1;
  /**********************************/
    hipEvent_t start, stop; 
	hipEventCreate(&start); //Creates an event object�
     hipEventCreate(&stop);
  /*************************************/
   // Declare array pointer ha and hb on Host/CPU
      int *ha;// input array
      int *hb;// output array
   //Allocate space for array pointed by ha on CPU
     ha=(int*)malloc(size*sizeof(int));
     hb=(int*)malloc(size*sizeof(int));
     //Check memory allocations on CPU
     if((ha==NULL)||(hb==NULL))
     {
	printf("\n Unable to allocate space on CPU for ha/hb ");
	exit(EXIT_FAILURE);
     }
  /**************************************/
   //Declare array pointer ga intended for device/gpu
     int *ga;
     //Allocate space for array pointed by ga on GPU
         err=hipMalloc((void **)&ga,size*sizeof(int));
	   //check memory allocation on GPU
	     if (hipSuccess!=err)
	     {
		printf("\n Memory allocation failed on GPU for ga");
		printf("\n error is- %s", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	     }
  /*******************************************************************/
   //Initialize ha
       for(i=0; i<size;i++)
	 {
	    ha[i]= (int) (rand()% 10);
	    //printf("\n%d", ha[i]);
	 }
  /*******************************************************************/
  //copy ha to ga from CPU to GPU
       if (hipSuccess!=hipMemcpy(ga,ha,size*sizeof(int),hipMemcpyHostToDevice))
	{
		printf("\n Error in copying ha to ga");
		exit(EXIT_FAILURE);
	}
  /*********************************************************************/
 /***********************************************************************/
  //Compute number of cuda blocks needed 
 
	blocks=(int)(size/blockSize);
	if ((size%blockSize)>0)
		blocks++;
	printf("\n The number of blocks needed=%d", blocks);
 /**********************************************************************/
       //launch the cuda kernel 
	hipEventRecord(start, 0); //Timestamp, zero �default stream
        SquareKernel<<<blocks,blockSize>>>(ga, size); 
        hipDeviceSynchronize();
		hipEventRecord(stop, 0); //Timestamp
        hipEventSynchronize(stop); 
		hipEventElapsedTime(&timespentGPU, start, stop); 
		printf("\n timespent on GPU=%f",timespentGPU);
  /*********************************************************************/
     //copy ga to ha from GPU to CPU
     if (hipSuccess!=hipMemcpy(hb,ga,size*sizeof(int),hipMemcpyDeviceToHost))
	{
		printf("\n Error in copying ga to hb");
		exit(EXIT_FAILURE);
	} 
 /**********************************************************************/
     // print values using hb
	 printf("\n values using GPU");
        for(i=0; i<size;i++)
          printf("\n%d", hb[i]);
  /**********************************************************************/
   //call the serial function SquareSerial
		start1=clock();
         SquareSerial(&ha[0],size);
		 stop1=clock();
		 timespentCPU = ((float)(stop1 - start1))/CLOCKS_PER_SEC;
		 printf("\n timespent on CPU=%f",timespentCPU);
		 getchar();
  /*********************************************************************/
    /**********************************************************************/
     // print values using ha
		 printf("\n values using CPU");
        for(i=0; i<size;i++)
          printf("\n%d", ha[i]);
  /**********************************************************************/
		printf("\n speed up=%f",(float)(timespentCPU/timespentGPU));

  /**********************************************************************/
    //Do clean up
	   free(ha);free(hb); //host pointers
	   hipFree(ga); // Device pointers
	   //Destroy events
	    hipEventDestroy(start); 
        hipEventDestroy(stop); 
     getchar();
     return 0;
}


__global__ void SquareKernel(int *ga, int size)
{
	int i, z;	
	 i=(blockIdx.x*blockDim.x)+threadIdx.x; //for multi block	
	if(i<size)
	{
	  z=ga[i];
      ga[i]=z*z;
	    
	}
}


//Serial C function to square
void SquareSerial(int* ha, int size)
{
	int i, j;
	
	for(i=0;i<size;i++)
	{
        j=ha[i];
		ha[i]=j*j;
	 
	}
}