//PROGRAM7
//THIS PROGRAM DEMONSTRATE USE OF PAGE LOCKED HOST MEMORY FOR SQUARING AN ARRAY USING A SIMPLE CUDA KERNEL
//Without measuring time
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void SquareKernel(int *,int);

int main()
{
  int i; //loop variable
  int blockSize=128, blocks; //for cuda blocks
  hipError_t err;//for error checking in cuda API
  int size=200;
  /**********************************/
   // Declare array pointer ha and allocate page locked host memory on CPU
      int *ha;// input array
	  if (hipSuccess!=hipHostAlloc(&ha, size*sizeof(int), hipHostMallocPortable))
     {
         printf("\n problem in allocating page locked memory");
         exit(EXIT_FAILURE);
     }   
   
  /**************************************/
   //Declare array pointer ga intended for device/gpu
     int *ga;
     //Allocate space for array pointed by ga on GPU
         err=hipMalloc((void **)&ga,size*sizeof(int));
	   //check memory allocation on GPU
	     if (hipSuccess!=err)
	     {
		   printf("\n Memory allocation failed on GPU for ga");
		   printf("\n error is- %s", hipGetErrorString(err));
		   exit(EXIT_FAILURE);
	     }
  /*******************************************************************/
   //Initialize ha
       for(i=0; i<size;i++)
	 {
	    ha[i]= (int) (rand()% 10);
	    //printf("\n%d", ha[i]);
	 }	  
  /*******************************************************************/
  //copy ha to ga from CPU to GPU
       if (hipSuccess!=hipMemcpy(ga,ha,size*sizeof(int),hipMemcpyHostToDevice))
	{
		printf("\n Error in copying ha to ga");
		exit(EXIT_FAILURE);
	}
  /*********************************************************************/
 /***********************************************************************/
  //Compute number of cuda blocks needed  
	blocks=(int)(size/blockSize);
	if ((size%blockSize)>0)
		blocks++;
	printf("\n The number of blocks needed=%d", blocks);
 /**********************************************************************/
       //launch the cuda kernel 
        SquareKernel<<<blocks,blockSize>>>(ga, size); 
        hipDeviceSynchronize();
  /*********************************************************************/
     //copy ga to ha from GPU to CPU
     if (hipSuccess!=hipMemcpy(ha,ga,size*sizeof(int),hipMemcpyDeviceToHost))
	{
		printf("\n Error in copying ga to hb");
		exit(EXIT_FAILURE);
	}
 
 /**********************************************************************/
     // print values using hb
	 printf("\n values using GPU");
        for(i=0; i<size;i++)
          printf("\n%d", ha[i]);
  /**********************************************************************/ 
    //Do clean up
	   hipHostFree(ha);//Free page locked memory pointers
	   hipFree(ga);// Device pointers
       getchar();
     return 0;
}


__global__ void SquareKernel(int *ga, int size)
{
	int i, z;	
	 i=(blockIdx.x*blockDim.x)+threadIdx.x; //for multi block	
	if(i<size)
	{
	  z=ga[i];
      ga[i]=z*z;
	    
	}
}


